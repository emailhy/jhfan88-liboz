#include "hip/hip_runtime.h"
//
// by Jan Eric Kyprianidis <www.kyprianidis.com>
// Copyright (C) 2010-2012 Computer Graphics Systems Group at the
// Hasso-Plattner-Institut, Potsdam, Germany <www.hpi3d.de>
//
// This program is free software: you can redistribute it and/or modify
// it under the terms of the GNU General Public License as published by
// the Free Software Foundation, either version 3 of the License, or
// (at your option) any later version.
//
// This program is distributed in the hope that it will be useful,
// but WITHOUT ANY WARRANTY; without even the implied warranty of
// MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
// GNU General Public License for more details.
//
#include <oz/stgauss2.h>
#include <oz/st.h>
#include <oz/st_util.h>
#include <oz/stintrk.h>
#include <oz/generate.h>
#include <oz/gpu_sampler2.h>
#include <oz/filter_gauss.h>

namespace {
    /*
    template <typename T, typename SRC>
    struct stgauss2_filter {
        const SRC& src_;
        float radius_;
        float twoSigma2_;
        T c_;
        float w_;

        inline __host__ __device__ stgauss2_filter( const SRC& src, float sigma )
             : src_(src)
         {
            radius_ = 2 * sigma;
            twoSigma2_ = 2 * sigma * sigma;
            c_ = make_zero<T>();
            w_ = 0;
        }

        inline __host__ __device__ float radius() const {
            return radius_;
        }

        inline __host__ __device__ void operator()(float sign, float u, float2 p) {
            #ifdef __CUDA_ARCH__
            float k = expf(-u * u / twoSigma2_);
            c_ += k * src_(p.x, p.y);
            w_ += k;
            #endif
        }
    };
    */


    template<typename T, int order, class SRC, class ST> struct StGauss2Filter : oz::generator<T> {
        unsigned w_, h_;
        const SRC src_;
        const ST st_;
        float sigma_, cos_max_;
        bool adaptive_;
        float step_size_;

        StGauss2Filter( unsigned w, unsigned h, const SRC& src, const ST& st,
                        float sigma, float cos_max, bool adaptive, float step_size )
            : w_(w), h_(h),
              src_(src), st_(st),
              sigma_(sigma), cos_max_(cos_max),
              adaptive_(adaptive), step_size_(step_size) {}

        inline __device__ T operator()( int ix, int iy ) const {
            float2 p0 = make_float2(ix + 0.5f, iy + 0.5f);
            float sigma = sigma_;
            if (adaptive_) {
                float A = oz::st2A(st_(p0.x, p0.y));
                sigma *= 0.25f * (1 + A)*(1 + A);
            }
            oz::filter_gauss_1d<T,SRC> f(src_, sigma);
            if (order == 1) oz::st_integrate_euler(p0, st_, f, cos_max_, w_, h_, step_size_);
            if (order == 2) oz::st_integrate_rk2(p0, st_, f, cos_max_, w_, h_, step_size_);
            if (order == 4) oz::st_integrate_rk4(p0, st_, f, cos_max_, w_, h_, step_size_);
            return f.result();
        }
    };


    template<typename T, int order>
    oz::gpu_image filterTO( const oz::gpu_image& src, bool src_linear,
                           const oz::gpu_image& st, bool st_linear,
                           float sigma, float max_angle, bool adaptive,
                           float step_size )
    {
        float cos_max = cosf(radians(max_angle));
        if (src.size() == st.size()) {
            return generate(src.size(), StGauss2Filter<T, order, gpu_sampler<T,0>, gpu_sampler<float3,1> >(
                src.w(), src.h(),
                gpu_sampler<T,0>(src, src_linear? hipFilterModeLinear : hipFilterModePoint),
                gpu_sampler<float3,1>(st, st_linear? hipFilterModeLinear : hipFilterModePoint),
                sigma, cos_max, adaptive, step_size));
        } else {
            float2 s = make_float2((float)st.w() / src.w(), (float)st.h() / src.h());
            return generate(src.size(), StGauss2Filter<T, order, gpu_sampler<T,0>, gpu_resampler<float3,1> >(
                src.w(), src.h(),
                gpu_sampler<T,0>(src, src_linear? hipFilterModeLinear : hipFilterModePoint),
                gpu_resampler<float3,1>(st, s, st_linear? hipFilterModeLinear : hipFilterModePoint),
                sigma, cos_max, adaptive, step_size));
        }
    }


    template<typename T>
    oz::gpu_image filterT( const oz::gpu_image& src, bool src_linear,
                          const oz::gpu_image& st, bool st_linear,
                          float sigma, float max_angle, bool adaptive,
                          int order, float step_size )
    {
        switch (order) {
            case 1: return filterTO<T,1>(src, src_linear, st, st_linear, sigma, max_angle, adaptive, step_size);
            case 2: return filterTO<T,2>(src, src_linear, st, st_linear, sigma, max_angle, adaptive, step_size);
            case 4: return filterTO<T,4>(src, src_linear, st, st_linear, sigma, max_angle, adaptive, step_size);
            default:
                OZ_X() << "Invalid order!";
        }
    }
}


oz::gpu_image oz::stgauss2_filter( const gpu_image& src, const gpu_image& st,
                                  float sigma, float max_angle, bool adaptive,
                                  bool src_linear, bool st_linear, int order, float step_size )
{
    if (sigma <= 0) return src;
    switch (src.format()) {
        case FMT_FLOAT:  return filterT<float >(src, src_linear, st, st_linear, sigma, max_angle, adaptive, order, step_size);
        case FMT_FLOAT3: return filterT<float3>(src, src_linear, st, st_linear, sigma, max_angle, adaptive, order, step_size);
        default:
            OZ_INVALID_FORMAT();
    }
}
